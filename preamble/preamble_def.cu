#include "hip/hip_runtime.h"
#ifndef PREAMBLE_DEFINE
#define PREAMBLE_DEFINE

#define sampler_t int

#define FLT_MIN 1.19209e-07
#define FLT_MAX 1e+37
#define MAX_ARRAY_SIZE 1000

#define uchar unsigned char
#define ushort unsigned short
#define uint unsigned int
#define ulong unsigned long

__device__ inline int2 operator+(int2 a, int2 b)
{
    return make_int2(a.x + b.x, a.y + b.y);
}

__device__ inline int4 operator+(int4 a, int4 b)
{
    return make_int4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
}

__device__ inline int2 operator*(int b, int2 a)
{
    return make_int2(b * a.x, b * a.y);
}

__device__ inline int4 operator*(int b, int4 a)
{
    return make_int4(b * a.x, b * a.y, b * a.z, b * a.w);
}

__device__ inline float pow(float x, int y) {
    return pow(float(x), float(y));
}

__device__ inline float2 sqrt(float2 a) {
    return make_float2(sqrt(a.x), sqrt(a.y));
}

__device__ inline float4 cross(float4 a, float4 b)
{ 
    return make_float4(a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x, 0); 
}

__device__ inline float dot(float4 a, float4 b)
{ 
    return a.x * b.x + a.y * b.y + a.z * b.z + a.w * b.w;
}

__device__ inline float length(float4 v)
{
    return sqrtf(dot(v, v));
}

__device__ inline unsigned int atomic_add(unsigned int* address, unsigned int value) {
    return atomicAdd(address, value);
}

__device__ inline uchar clij_convert_uchar_sat(float value) {
    if (value > 255) {
        return 255;
    }
    if (value < 0) {
        return 0;
    }
    return (uchar)value;
}

__device__ inline char clij_convert_char_sat(float value) {
    if (value > 127) {
        return 127;
    }
    if (value < -128) {
        return -128;
    }
    return (char)value;
}

__device__ inline ushort clij_convert_ushort_sat(float value) {
    if (value > 65535) {
        return 65535;
    }
    if (value < 0) {
        return 0;
    }
    return (ushort)value;
}

__device__ inline short clij_convert_short_sat(float value) {
    if (value > 32767) {
        return 32767;
    }
    if (value < -32768) {
        return -32768;
    }
    return (short)value;
}

__device__ inline uint clij_convert_uint_sat(float value) {
    if (value > 4294967295) {
        return 4294967295;
    }
    if (value < 0) {
        return 0;
    }
    return (uint)value;
}

__device__ inline uint convert_uint_sat(float value) {
    if (value > 4294967295) {
        return 4294967295;
    }
    if (value < 0) {
        return 0;
    }
    return (uint)value;
}

__device__ inline int clij_convert_int_sat(float value) {
    if (value > 2147483647) {
        return 2147483647;
    }
    if (value < -2147483648) {
        return -2147483648;
    }
    return (int)value;
}

__device__ inline uint clij_convert_ulong_sat(float value) {
    if (value > 18446744073709551615) {
        return 18446744073709551615;
    }
    if (value < 0) {
        return 0;
    }
    return (ulong)value;
}

__device__ inline int clij_convert_long_sat(float value) {
    if (value > 9223372036854775807) {
        return 9223372036854775807;
    }
    if (value < -9223372036854775808 ) {
        return -9223372036854775808 ;
    }
    return (long)value;
}

__device__ inline float clij_convert_float_sat(float value) {
    return value;
}

#define READ_IMAGE(a,b,c) READ_ ## a ## _IMAGE(a,b,c)
#define WRITE_IMAGE(a,b,c) WRITE_ ## a ## _IMAGE(a,b,c)

#define GET_IMAGE_WIDTH(image_key) IMAGE_SIZE_ ## image_key ## _WIDTH
#define GET_IMAGE_HEIGHT(image_key) IMAGE_SIZE_ ## image_key ## _HEIGHT
#define GET_IMAGE_DEPTH(image_key) IMAGE_SIZE_ ## image_key ## _DEPTH

#define CLK_NORMALIZED_COORDS_FALSE 1
#define CLK_ADDRESS_CLAMP_TO_EDGE 2
#define CLK_FILTER_NEAREST 4
#define CLK_NORMALIZED_COORDS_TRUE 8
#define CLK_ADDRESS_CLAMP 16
#define CLK_FILTER_LINEAR 32
#define CLK_ADDRESS_NONE 64

#endif // PREAMBLE_DEFINE