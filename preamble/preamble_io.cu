
#include <hip/hip_runtime.h>
#ifdef BUFFER_READ_WRITE
#define BUFFER_READ_WRITE
#define MINMAX_TYPE int

__device__ inline {pixel_type}2 read_buffer3d{short_pixel_type}(int read_buffer_width, int read_buffer_height, int read_buffer_depth, {pixel_type} * buffer_var, int sampler, int4 position )
{
    int4 pos = make_int4(position.x, position.y, position.z, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width + pos.z * read_buffer_width * read_buffer_height;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height || pos.z < 0 || pos.z >= read_buffer_depth) {
        return make_{pixel_type}2(0, 0);
    }
    return make_{pixel_type}2(buffer_var[pos_in_buffer],0);
}

__device__ inline void write_buffer3d{short_pixel_type}(int write_buffer_width, int write_buffer_height, int write_buffer_depth, {pixel_type} * buffer_var, int4 pos, float value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width + pos.z * write_buffer_width * write_buffer_height;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height || pos.z < 0 || pos.z >= write_buffer_depth) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline {pixel_type}2 read_buffer2d{short_pixel_type}(int read_buffer_width, int read_buffer_height, int read_buffer_depth, {pixel_type} * buffer_var, int sampler, int2 position )
{
    int4 pos = make_int4(position.x, position.y, 0, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height) {
        return make_{pixel_type}2(0, 0);
    }
    return make_{pixel_type}2(buffer_var[pos_in_buffer],0);
}

__device__ inline void write_buffer2d{short_pixel_type}(int write_buffer_width, int write_buffer_height, int write_buffer_depth, {pixel_type} * buffer_var, int2 pos, float value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline {pixel_type}2 read_buffer1d{short_pixel_type}(int read_buffer_width, int read_buffer_height, int read_buffer_depth, {pixel_type} * buffer_var, int sampler, int position )
{
    int pos = position;

    pos = max((MINMAX_TYPE)pos, (MINMAX_TYPE)0);
    pos = min((MINMAX_TYPE)pos, (MINMAX_TYPE)read_buffer_width - 1);

    int pos_in_buffer = pos;
    if (pos < 0 || pos >= read_buffer_width) {
        return make_{pixel_type}2(0, 0);
    }
    return make_{pixel_type}2(buffer_var[pos_in_buffer],0);
}

__device__ inline void write_buffer1d{short_pixel_type}(int write_buffer_width, int write_buffer_height, int write_buffer_depth, {pixel_type} * buffer_var, int pos, ulong value )
{
    int pos_in_buffer = pos;
    if (pos < 0 || pos >= write_buffer_width) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

#endif // BUFFER_READ_WRITE